#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "hipDNN.h"

#ifdef __cplusplus
extern "C" {
#endif
#include "main_kernel.h"
#ifdef __cplusplus
}
#endif


// Error handling
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if ( hipSuccess != err) {
        fprintf(stderr,
            "CUDA Driver API error = %04d from file <%s>, line %i.\n",
            err, file, line);
        exit(-1);
    }
}


__global__ void testKernel(float *data, const int l)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if (idx < l)
        data[idx]++;
}

#ifdef __cplusplus
extern "C" {
#endif
void test_gpu(float *data, const int length)
{
    // Run kernel
    testKernel<<< 512, 1024 >>>(data, length);
}
#ifdef __cplusplus
}
#endif
