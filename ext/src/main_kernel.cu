#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#ifdef __cplusplus
extern "C" {
#endif
#include "main_kernel.h"
#ifdef __cplusplus
}
#endif


// Error handling
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if ( hipSuccess != err) {
        fprintf(stderr,
            "CUDA Driver API error = %04d from file <%s>, line %i.\n",
            err, file, line);
        exit(-1);
    }
}


__global__ void testKernel(float *data, const int l)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    data[idx]++;
}

#ifdef __cplusplus
extern "C" {
#endif
void test_gpu(float *h_data, const int length)
{
    float *d_data;
    int b_length = length * sizeof(int);
    checkCudaErrors(hipMalloc(&d_data, b_length));
    
    checkCudaErrors(hipMemcpy(d_data, h_data, b_length, hipMemcpyHostToDevice));
    
    // Run kernel
    testKernel<<< length, 1 >>>(d_data, length);

    checkCudaErrors(hipMemcpy(h_data, d_data, b_length, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_data));
}
#ifdef __cplusplus
}
#endif
