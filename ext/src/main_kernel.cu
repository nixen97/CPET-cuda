#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "hipDNN.h"

#ifdef __cplusplus
extern "C" {
#endif
#include "main_kernel.h"
#ifdef __cplusplus
}
#endif


// Error handling
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if ( hipSuccess != err) {
        fprintf(stderr,
            "CUDA Driver API error = %04d from file <%s>, line %i.\n",
            err, file, line);
        exit(-1);
    }
}


__global__ void testKernel(float *data, const int l)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if (idx < l)
        data[idx]++;
}

#ifdef __cplusplus
extern "C" {
#endif
void test_gpu(float *data, const int length)
{
    // Run kernel
    testKernel<<< 512, 1024 >>>(data, length);
}
#ifdef __cplusplus
}
#endif

// void test_gpu(const float *data, const float *result, const int length)
// {
//     // Allocate memory on device
//     void *d_data, *d_result;
    
//     checkCudaErrors(hipMalloc(&d_data, length));
//     checkCudaErrors(hipMalloc(&d_result, length));

//     // Copy data to device
//     checkCudaErrors(hipMemcpy(d_data, data, length, hipMemcpyHostToDevice));

//     // Run kernel
//     testKernel<<< 512, 1024 >>>(d_data, d_result, length);

//     // Copy result back
//     checkCudaErrors(hipMemcpy(result, d_result, length, hipMemcpyDeviceToHost));

//     // Free device memory
//     checkCudaErrors(hipFree(d_data));
//     checkCudaErrors(hipFree(d_result));
// }