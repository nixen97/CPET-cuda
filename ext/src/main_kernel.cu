#include "hip/hip_runtime.h"
#include "main_kernel.h"

__global__ void testKernel(const float *data, const float *result, const int l)
{
    index = blockIdx.x*blockDim.x+threadIdx.x;
    if (index < l)
        result[index] = data[index]++;
}

void test_gpu(const float *data, const float *result, const int length)
{
    // Allocate memory on device
    float *d_data, *d_result
    
    checkCudaErrors(hipMalloc(&d_data, length));
    checkCudaErrors(hipMalloc(&d_result, length));

    // Copy data to device
    checkCudaErrors(hipMemcpy(d_data, data, length, hipMemcpyHostToDevice));

    // Run kernel
    testKernel<<< 512, 1024 >>>(d_data, d_result, length);

    // Copy result back
    checkCudaErrors(hipMemcpy(result, d_result, length, hipMemcpyDeviceToHost));

    // Free device memory
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_result));
}